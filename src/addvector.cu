#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int *a, *b;  // host data
int *c, *c2;  // results
int *a_d,*b_d,*c_d;
//int n=10000000;
int n = 100 * 100;
int block_size, block_no; 

char retdata[50];


// detectEdge(int *imgIn, *imgOut)
// add up color of all neighbors
// get avg
// compare to color of this pixel
// if its greater than threshold
// return color set
// else return 0

__global__ void detectEdge(int *imgIn,int *B,int *imgOut,int N)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int y = i / 100;
   int x = i % 100;
   int sum = 0;
   for (int oy=-1; oy<=1; oy++) {
	 for (int ox=-1; ox<=1; ox++) {
	   int index = (y+oy) * 100 + (x+ox);
	   sum += imgIn[index];
	 }
   }
   float avg = sum / 9.0;
   float diff = imgIn[i] - avg;
   if (abs(diff) > 1000) imgOut[i] = 255*255*100;
   else imgOut[i] = 0;
}

__global__ void vecAdd(int *A,int *B,int *C,int N)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   C[i] = A[i] + B[i]; 
}

void vecAdd_h(int *A1,int *B1, int *C1, int N)
{
   for(int i=0;i<N;i++)
	  C1[i] = A1[i] * B1[i];
}

extern "C" { 

	void init()
	{
	   printf("Begin \n");
	   int nBytes = n*sizeof(int);
	   a = (int *)malloc(nBytes);
	   b = (int *)malloc(nBytes);
	   c = (int *)malloc(nBytes);
	   c2 = (int *)malloc(nBytes);
	   block_size=4000;
	   block_no = n/block_size;
	   dim3 dimBlock(block_size,1,1);
	   dim3 dimGrid(block_no,1,1);
	   for(int i=0;i<n;i++)
		  a[i]=i,b[i]=i;
	   printf("Allocating device memory on host..\n");
	   hipMalloc((void **)&a_d,n*sizeof(int));
	   hipMalloc((void **)&b_d,n*sizeof(int));
	   hipMalloc((void **)&c_d,n*sizeof(int));
	}
	
	int* calc(int64_t **img) {
	   printf("Copying to device..\n");
	   hipMemcpy(a_d,a,n*sizeof(int),hipMemcpyHostToDevice);
	   hipMemcpy(a_d,img,n*sizeof(int),hipMemcpyHostToDevice);
	   hipMemcpy(b_d,b,n*sizeof(int),hipMemcpyHostToDevice);
	   clock_t start_d=clock();
	   printf("Doing GPU edge detect\n");
	   detectEdge<<<block_no,block_size>>>(a_d,b_d,c_d,n);
	   hipDeviceSynchronize();
	   clock_t end_d = clock();
	   double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
	   hipMemcpy(c,c_d,n*sizeof(int),hipMemcpyDeviceToHost);
	   return c;
	}
	
	void done() {
	   hipFree(a_d);
	   hipFree(b_d);
	   hipFree(c_d);
	}

}
